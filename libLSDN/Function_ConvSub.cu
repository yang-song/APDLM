#include "hip/hip_runtime.h"
//Author: Alexander G. Schwing (http://alexander-schwing.de)
#ifdef _MSC_VER
#pragma warning( disable : 4661 )
#endif
#include "Function_ConvSub.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "../LSDN_CudaCommon.h"
#include "LSDN_mathfunctions.h"

template <typename T>
__global__ void kernel_AccumPatchDiff2ImSubsample(const int num, T* img, int numRow, int numCol, int numChan,
	int kNumRow, int kNumCol, int stride, int pad, const T* patchMatrixDiff, int h_out, int w_out, int subsample_h, int subsample_w) {
	CUDA_KERNEL_LOOP(index, num) {//loops over channels*numImageRows*numImageCols
		T res = 0;
		//coordinates in padded grid
		int h = index % numRow + pad;// +(kNumRow - 1)*(subsample_h / 2) - (subsample_h > 1)*pad;
		int w = (index / numRow) % numCol + pad;// +(kNumCol - 1)*(subsample_w / 2) - (subsample_w > 1)*pad;
		int c = index / (numRow * numCol);

		//where did pixel (index%numRow, (index/numRow)%numCol) have an influence in the patchmatrix?
		//the ranges in patchMatrixDiff that affect the current position (h,w) in img
		int h_out_start = (h < kNumRow*subsample_h) ? h%subsample_h : (h - kNumRow*subsample_h) / stride + subsample_h;
		int h_out_end = min(h / stride + 1, h_out);
		int w_out_start = (w < kNumCol*subsample_w) ? w%subsample_w : (w - kNumCol*subsample_w) / stride + subsample_w;
		int w_out_end = min(w / stride + 1, w_out);

		
		for (int w_c = w_out_start; w_c < w_out_end; w_c+=subsample_w) {
			for (int h_c = h_out_start; h_c < h_out_end; h_c+=subsample_h) {//height offset for grid in padded dimensions
				int c_out = c * kNumRow * kNumCol + ((w - w_c * stride)/subsample_w) * kNumRow + (h - h_c * stride)/subsample_h;
				res += patchMatrixDiff[(c_out * w_out + w_c) * h_out + h_c];
			}
		}
		//printf("%f ", patchMatrixDiff[0]);
		
		/*//equivalent implementation, fewer multiplications within for loops
		int offset = (c * kNumRow * kNumCol + w * kNumRow + h) * w_out * h_out;
		int coeff_h_out = (1 - stride * w_out * h_out);
		int coeff_w_out = (1 - stride * kNumRow * w_out) * h_out;

		for (int w_c = w_out_start; w_c < w_out_end; ++w_c) {
			for (int h_c = h_out_start; h_c < h_out_end; ++h_c) {
				res += patchMatrixDiff[offset + w_c*coeff_w_out + h_c*coeff_h_out];
			}
		}
		*/
		img[index] = res;

	}
}

template <class N>
void ConvSubFunction<N>::AccumPatchDiff2Im(i2t<true>, ValueType* img, SizeType numRow, SizeType numCol, SizeType numChan, SizeType kNumRow, SizeType kNumCol) {
	//accumulated the gradients of extracted patches back to img

	//SizeType h_out = (numRow + 2 * padSize_ - kNumRow*SubsampleH_ + ((SubsampleH_>1) ? 1 : 0)) / stride_ + 1;
	//SizeType w_out = (numCol + 2 * padSize_ - kNumCol*SubsampleW_ + ((SubsampleW_>1) ? 1 : 0)) / stride_ + 1;
	SizeType h_out = (numRow + 2 * padSize_ - ((kNumRow - 1)*SubsampleH_ + 1)) / stride_ + 1;
	SizeType w_out = (numCol + 2 * padSize_ - ((kNumCol - 1)*SubsampleW_ + 1)) / stride_ + 1;
	SizeType num_kernels = numChan * numRow * numCol;

	// To avoid involving atomic operations, we launch one kernel per
	// input dimension, and then in the kernel add up the output dimensions.
	kernel_AccumPatchDiff2ImSubsample<ValueType> << <LSDN_GET_BLOCKS(num_kernels), LSDN_CUDA_NUM_THREADS >> >(int(num_kernels), img,
		int(numRow), int(numCol), int(numChan), int(kNumRow), int(kNumCol), int(stride_), int(padSize_),
		patchMatrixDiff_, int(h_out), int(w_out), int(SubsampleH_), int(SubsampleW_));

	check_cuda_errors(__FILE__, __LINE__);
}

template <typename T>
__global__ void kernel_Im2PatchesSubsample(const int num, const T* img, int numRow, int numCol,
	int kNumRow, int kNumCol, int stride, int pad, T* patchMatrix, int h_out, int w_out, int subsample_h, int subsample_w) {
	CUDA_KERNEL_LOOP(index, num) {//loops over channels*h_out*w_out
		int h_d = index % h_out;
		index /= h_out;
		int w_d = index % w_out;//width-destination
		int c_s = index / w_out;//channel-source
		int c_d = c_s * kNumRow * kNumCol;//channel-destination offset
		int h_s = h_d * stride - pad;// -(kNumRow - 1)*(subsample_h / 2) + (subsample_h > 1)*pad;
		int w_s = w_d * stride - pad;// -(kNumCol - 1)*(subsample_w / 2) + (subsample_w > 1)*pad;
		patchMatrix += (c_d * w_out + w_d) * h_out + h_d;
		img += (c_s * numCol + w_s) * numRow + h_s;

		for (int c = 0; c < kNumCol; ++c) {
			for (int r = 0; r < kNumRow; ++r) {
				int h = h_s + r*subsample_h;
				int w = w_s + c*subsample_w;

				*patchMatrix = (h >= 0 && h<numRow && w >= 0 && w<numCol) ? img[c*subsample_w*numRow + r*subsample_h] : 0;
				patchMatrix += w_out * h_out;
			}
		}

	}
}

template <class N>
void ConvSubFunction<N>::Im2Patches(i2t<true>, ValueType* img, SizeType numRow, SizeType numCol, SizeType numChan, SizeType kNumRow, SizeType kNumCol) {
	//extract patches of size kNumRow-kNumCol-numChannel from img, and concatenate them into a matrix
	//each row stores the column vectorized of each patch

	//each kernel copies a single-channel grid (i.e., a kNumRow-kNumCol region)
	//SizeType h_out = (numRow + 2 * padSize_ - kNumRow*SubsampleH_ + ((SubsampleH_>1)?1:0)) / stride_ + 1;
	//SizeType w_out = (numCol + 2 * padSize_ - kNumCol*SubsampleW_ + ((SubsampleW_>1)?1:0)) / stride_ + 1;
	SizeType h_out = (numRow + 2 * padSize_ - ((kNumRow - 1)*SubsampleH_ + 1)) / stride_ + 1;
	SizeType w_out = (numCol + 2 * padSize_ - ((kNumCol - 1)*SubsampleW_ + 1)) / stride_ + 1;
	SizeType num_kernels = numChan * h_out * w_out;

	check_cuda_errors(__FILE__, __LINE__);
	kernel_Im2PatchesSubsample<ValueType> << <LSDN_GET_BLOCKS(num_kernels), LSDN_CUDA_NUM_THREADS >> >(int(num_kernels), img,
		int(numRow), int(numCol), int(kNumRow), int(kNumCol), int(stride_), int(padSize_),
		patchMatrix_, int(h_out), int(w_out), int(SubsampleH_), int(SubsampleW_));
	check_cuda_errors(__FILE__, __LINE__);
}

template <typename T>
__global__ void kernel_AdditionModuloOperand(T* res, int numEl, const T* addend, int op_division, int op_modulo) {
	CUDA_KERNEL_LOOP(index, numEl) {
		int ix = (index / op_division) % op_modulo;
		res[index] += addend[ix];
	}
}

template <class N>
void ConvSubFunction<N>::AdditionModuloOperand(i2t<true>, ValueType* res, SizeType numEl, ValueType* addend, SizeType op_division, SizeType op_modulo) {
	kernel_AdditionModuloOperand<ValueType> << <LSDN_GET_BLOCKS(numEl), LSDN_CUDA_NUM_THREADS >> >(res, numEl, addend, op_division, op_modulo);
	check_cuda_errors(__FILE__, __LINE__);
}

template <typename T>
__global__ void kernel_BiasDerivativeSingleDim(T* res, const T* input, const int patchSize, const int numSamples, const int numChannels, const int sampleSize, bool performAddition) {
	if (performAddition) {
		CUDA_KERNEL_LOOP(index, numChannels) {
			const T* ptr = input + patchSize*index;
			for (int k = 0; k < patchSize*numSamples; ++k) {
				int sample = k / patchSize;
				int offset = k % patchSize;
				res[index] += ptr[sampleSize*sample + offset];
			}
		}
	} else {
		CUDA_KERNEL_LOOP(index, numChannels) {
			const T* ptr = input + patchSize*index;
			res[index] = T(0);
			for (int k = 0; k < patchSize*numSamples; ++k) {
				int sample = k / patchSize;
				int offset = k % patchSize;
				res[index] += ptr[sampleSize*sample + offset];
			}
		}
	}
}

template <class N>
void ConvSubFunction<N>::BiasDerivativeSingleDim(i2t<true>, ValueType* res, ValueType* input, SizeType patchSize, SizeType numSamples, SizeType numChannels, bool performAddition) {
	kernel_BiasDerivativeSingleDim<ValueType> << <LSDN_GET_BLOCKS(numChannels), LSDN_CUDA_NUM_THREADS >> >(res, input, patchSize, numSamples, numChannels, patchSize*numChannels, performAddition);
	check_cuda_errors(__FILE__, __LINE__);
}

template <typename T>
__global__ void kernel_BiasDerivativeMultiDim(T* res, const T* input, const int sampleSize, const int numSamples, bool performAddition) {
	if (performAddition) {
		CUDA_KERNEL_LOOP(index, sampleSize) {
			for (int k = 0; k < numSamples; ++k) {
				res[index] += input[k*sampleSize + index];
			}
		}
	} else {
		CUDA_KERNEL_LOOP(index, sampleSize) {
			res[index] = T(0);
			for (int k = 0; k < numSamples; ++k) {
				res[index] += input[k*sampleSize + index];
			}
		}
	}
}

template <class N>
void ConvSubFunction<N>::BiasDerivativeMultiDim(i2t<true>, ValueType* res, ValueType* input, SizeType sampleSize, SizeType numSamples, bool performAddition) {
	kernel_BiasDerivativeMultiDim<ValueType> << <LSDN_GET_BLOCKS(sampleSize), LSDN_CUDA_NUM_THREADS >> >(res, input, sampleSize, numSamples, performAddition);
	check_cuda_errors(__FILE__, __LINE__);
}

template class ConvSubFunction<Node<double, int, false> >;
template class ConvSubFunction<Node<double, int, true> >;
template class ConvSubFunction<Node<float, int, false> >;
template class ConvSubFunction<Node<float, int, true> >;
